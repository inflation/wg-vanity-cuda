#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "vanity.h"

__global__ void test_vanity() {
  vanity(reinterpret_cast<const uint8_t *>("asuna"), 5);
}

int main() {
  int blockSize, minGridSize, maxActiveBlocks, gridSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, test_vanity);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, test_vanity,
                                                blockSize, 0);
  printf("blockSize: %d, minGridSize: %d, maxActiveBlocks: %d\n", blockSize,
         minGridSize, maxActiveBlocks);

  int device;
  hipDeviceProp_t props;
  hipGetDevice(&device);
  hipGetDeviceProperties(&props, device);

  float occupancy = ((float)maxActiveBlocks * blockSize / props.warpSize) /
                    ((float)props.maxThreadsPerMultiProcessor / props.warpSize);

  gridSize = minGridSize * 1024;
  test_vanity<<<gridSize, blockSize>>>();
  printf("Launched blocks of size %d. Theoretical occupancy: %f\n", blockSize,
         occupancy);

  hipDeviceReset();
  return 0;
}