#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "common.h"
#include "vanity.h"

__device__ uint8_t vanity_str[KEY_LEN_BASE64];

__global__ void test_vanity(size_t len) {
  vanity(vanity_str, len);
}

int main(int argc, char** argv) {
  char* str = argv[1];
  size_t len = strlen(str);
  printf("String: %s, Length: %zu\n", str, len);

  checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(vanity_str), str, len));

  int blockSize, minGridSize, maxActiveBlocks, gridSize;
  checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
                                                    test_vanity));
  checkCudaError(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &maxActiveBlocks, test_vanity, blockSize, 0));
  printf("blockSize: %d, minGridSize: %d, maxActiveBlocks: %d\n", blockSize,
         minGridSize, maxActiveBlocks);

  int device;
  hipDeviceProp_t props;
  checkCudaError(hipGetDevice(&device));
  checkCudaError(hipGetDeviceProperties(&props, device));

  float occupancy = ((float)maxActiveBlocks * blockSize / props.warpSize) /
                    ((float)props.maxThreadsPerMultiProcessor / props.warpSize);

  gridSize = minGridSize * 1024;
  test_vanity<<<gridSize, blockSize>>>(len);
  checkLastCudaError();

  std::cout << "Launched blocks of size: " << blockSize
            << ". Theoretical occupancy: " << occupancy << std::endl;

  checkCudaError(hipDeviceReset());
  return 0;
}